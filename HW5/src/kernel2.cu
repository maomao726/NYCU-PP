#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define TILE_WIDTH 8

__global__ void mandelKernel( int* d_data,
                              int width,
                              float lowerX, float lowerY,
                              float stepX, float stepY,
                              int max_iteration, int pitch)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;

    float z_re = c_re, z_im = c_im;
    int i;
    for(i = 0; i < max_iteration; i++)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    //write i back
    int* target = (int*)((char*)d_data + thisY * pitch) + thisX;
    *target = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int img_size = resX * resY * sizeof(int);
    int* h_data;
    int* d_data;
    size_t pitch;
    
    //kernel config/invoke
    hipHostAlloc((void**) &h_data, img_size, hipHostMallocDefault);
    hipMallocPitch((void**) &d_data, &pitch, resX * sizeof(int), resY);
    dim3 dimGrid(resX / TILE_WIDTH, resY / TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    mandelKernel<<<dimGrid, dimBlock>>>(d_data, resX, lowerX, lowerY, stepX, stepY, maxIterations, pitch);

    hipMemcpy2D(h_data, resX * sizeof(int), d_data, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_data, img_size);

    hipFree(d_data);
    hipHostFree(h_data);
}
